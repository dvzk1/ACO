#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// PARÂMETROS
// se for executar outras matrizes, mudar o valor de cidades para a quantidade de cidades da matriz
// estamos executando para 100x100 (100 cidades)

#define CIDADES 100 // definir tam. da matriz

// parametros standard
#define FORMIGAS 4000
#define DIST_MAX 150
#define ALFA 1
#define BETA 5
#define EVA 0.5
#define QTD_FEROMONIO 100
#define MAX_TOURS 50
#define FEROMONIO_INICIAL (1.0/CIDADES)
#define DIST_TOTAL_MAXIMA (CIDADES * DIST_MAX)

// FORMIGA
struct formiga {
    int cidadeAtual, proxCidade, indiceCaminho;
    int visitado[CIDADES];
    int caminho[CIDADES];
    float comprimentoTour;
};

// Variáveis globais na CPU
float distancias[CIDADES][CIDADES];
float feromonios[CIDADES][CIDADES];
struct formiga formigas[FORMIGAS];
float melhorDistancia = (float)DIST_TOTAL_MAXIMA;

// Declaração de funções CUDA
__global__ void inicializarFormigasCUDA(struct formiga *formigas, int numFormigas, int numCidades, unsigned int seed, hiprandState *states);
__global__ void moverFormigasCUDA(struct formiga *formigas, float *distancias, float *feromonios, int numFormigas, int numCidades, float feromonioInicial, float evap, float qntdFeromonio, int maxTours, float *melhorDistancia, hiprandState *states);
__device__ int proximaCidadeCUDA(struct formiga *formiga, float *distancias, float *feromonios, int numCidades, hiprandState *state);

// Verificando se a entrada está dentro dos limites permitidos
void obterMatrizDistancias() {
    int i, j;
    float k;
    while (scanf("%i %i %f", &i, &j, &k) == 3) {
        if (i < CIDADES && j < CIDADES && k <= DIST_MAX) {
            distancias[i][j] = k;
            distancias[j][i] = k; 
            feromonios[i][j] = FEROMONIO_INICIAL;
            feromonios[j][i] = FEROMONIO_INICIAL;
        } else {
            printf("Erro: Indices fora do limite ou distancia maior que o permitido.\n");
            exit(1);
        }
    }
}

// MAIN
int main() {
    clock_t inicio, fim;
    double tempoGasto;

    // Registrar o tempo de início
    inicio = clock();

    // Obter a matriz de distâncias
    obterMatrizDistancias();

    // Alocar memória na GPU
    struct formiga *d_formigas;
    float *d_distancias;
    float *d_feromonios;
    float *d_melhorDistancia;
    hiprandState *d_states;
    hipMalloc((void**)&d_formigas, FORMIGAS * sizeof(struct formiga));
    hipMalloc((void**)&d_distancias, CIDADES * CIDADES * sizeof(float));
    hipMalloc((void**)&d_feromonios, CIDADES * CIDADES * sizeof(float));
    hipMalloc((void**)&d_melhorDistancia, sizeof(float));
    hipMalloc((void**)&d_states, FORMIGAS * sizeof(hiprandState));

    // Copiar dados para a GPU
    hipMemcpy(d_distancias, distancias, CIDADES * CIDADES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_feromonios, feromonios, CIDADES * CIDADES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_melhorDistancia, &melhorDistancia, sizeof(float), hipMemcpyHostToDevice);

    // Inicializar formigas na GPU
    inicializarFormigasCUDA<<<(FORMIGAS + 255) / 256, 256>>>(d_formigas, FORMIGAS, CIDADES, time(NULL), d_states);
    hipDeviceSynchronize(); // Certificar-se de que as formigas foram inicializadas

    // Mover formigas na GPU
    moverFormigasCUDA<<<(FORMIGAS + 255) / 256, 256>>>(d_formigas, d_distancias, d_feromonios, FORMIGAS, CIDADES, FEROMONIO_INICIAL, EVA, QTD_FEROMONIO, MAX_TOURS, d_melhorDistancia, d_states);
    hipDeviceSynchronize(); // Certificar-se de que a movimentação das formigas terminou

    // Copiar resultado de volta para a CPU
    hipMemcpy(&melhorDistancia, d_melhorDistancia, sizeof(float), hipMemcpyDeviceToHost);

    // Copiar formigas de volta para a CPU para verificação
    hipMemcpy(formigas, d_formigas, FORMIGAS * sizeof(struct formiga), hipMemcpyDeviceToHost);

/*
    // Imprimir alguns exemplos das variáveis na CPU
    for (int i = 0; i < 5; ++i) {
        printf("Formiga %d: Comprimento do Tour: %f\n", i, formigas[i].comprimentoTour);
        printf("Caminho: ");
        for (int j = 0; j < CIDADES; ++j) {
            if (formigas[i].caminho[j] != -1)
                printf("%d ", formigas[i].caminho[j]);
        }
        printf("\nVisitado: ");
        for (int j = 0; j < CIDADES; ++j) {
            printf("%d ", formigas[i].visitado[j]);
        }
        printf("\n");
    }
*/

    // Liberar memória da GPU
    hipFree(d_formigas);
    hipFree(d_distancias);
    hipFree(d_feromonios);
    hipFree(d_melhorDistancia);
    hipFree(d_states);

    // Registrar o tempo de fim
    fim = clock();

    // Calcular o tempo gasto
    tempoGasto = ((double)(fim - inicio)) / CLOCKS_PER_SEC;

    printf("Melhor distância: %f\n", melhorDistancia);
    printf("Tempo para gerar: %f segundos\n", tempoGasto);

    return 0;
}

// Func. CUDA

// Função para inicializar as formigas
__global__ void inicializarFormigasCUDA(struct formiga *formigas, int numFormigas, int numCidades, unsigned int seed, hiprandState *states) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < numFormigas) {
        hiprand_init(seed, id, 0, &states[id]); 
        struct formiga *formiga = &formigas[id];
        formiga->cidadeAtual = hiprand(&states[id]) % numCidades; 
        formiga->indiceCaminho = 1;
        formiga->comprimentoTour = 0.0f;
        for (int i = 0; i < numCidades; ++i) {
            formiga->visitado[i] = 0;
            formiga->caminho[i] = -1;
        }
        formiga->visitado[formiga->cidadeAtual] = 1; // Marcar a cidade inicial como visitada
        formiga->caminho[0] = formiga->cidadeAtual; // Adicionar a cidade inicial ao caminho

        printf("Formiga %d inicializada na cidade %d\n", id, formiga->cidadeAtual);
    }
}

// Função para mover as formigas
__global__ void moverFormigasCUDA(struct formiga *formigas, float *distancias, float *feromonios, int numFormigas, int numCidades, float feromonioInicial, float evap, float qntdFeromonio, int maxTours, float *melhorDistancia, hiprandState *states) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < numFormigas) {
        hiprandState state = states[id];
        struct formiga *formiga = &formigas[id];
        for (int tour = 0; tour < maxTours; ++tour) {
            formiga->indiceCaminho = 1;
            formiga->comprimentoTour = 0.0f;
            for (int i = 0; i < numCidades; ++i) {
                formiga->visitado[i] = 0;
                formiga->caminho[i] = -1;
            }
            formiga->cidadeAtual = hiprand(&state) % numCidades; // Selecionar uma cidade inicial aleatória para cada tour
            formiga->visitado[formiga->cidadeAtual] = 1;
            formiga->caminho[0] = formiga->cidadeAtual;

            // Mover a formiga até que todas as cidades sejam visitadas
            while (formiga->indiceCaminho < numCidades) {
                int proxima = proximaCidadeCUDA(formiga, distancias, feromonios, numCidades, &state);
                formiga->visitado[proxima] = 1;
                formiga->caminho[formiga->indiceCaminho++] = proxima;
                formiga->comprimentoTour += distancias[formiga->cidadeAtual * numCidades + proxima];
                formiga->cidadeAtual = proxima;
            }
            formiga->comprimentoTour += distancias[formiga->cidadeAtual * numCidades + formiga->caminho[0]];

            printf("Formiga %d completou um tour com comprimento %f\n", id, formiga->comprimentoTour);

            // Evaporação dos feromônios
            for (int i = 0; i < numCidades; ++i) {
                for (int j = 0; j < numCidades; ++j) {
                    atomicExch(&feromonios[i * numCidades + j], (1.0 - evap) * feromonios[i * numCidades + j]);
                    if (feromonios[i * numCidades + j] < feromonioInicial) {
                        atomicExch(&feromonios[i * numCidades + j], feromonioInicial);
                    }
                }
            }

            // Atualização dos feromônios com base no tour da formiga
            for (int i = 0; i < numCidades - 1; ++i) {
                int de = formiga->caminho[i];
                int para = formiga->caminho[i + 1];
                atomicAdd(&feromonios[de * numCidades + para], qntdFeromonio / formiga->comprimentoTour);
                atomicAdd(&feromonios[para * numCidades + de], qntdFeromonio / formiga->comprimentoTour);
            }
            int de = formiga->caminho[numCidades - 1];
            int para = formiga->caminho[0];
            atomicAdd(&feromonios[de * numCidades + para], qntdFeromonio / formiga->comprimentoTour);
            atomicAdd(&feromonios[para * numCidades + de], qntdFeromonio / formiga->comprimentoTour);

            // Atualizar a melhor distância encontrada
            atomicMin((int *)melhorDistancia, __float_as_int(formiga->comprimentoTour));
        }
        states[id] = state;
    }
}

// Função para determinar a próxima cidade a ser visitada pela formiga
__device__ int proximaCidadeCUDA(struct formiga *formiga, float *distancias, float *feromonios, int numCidades, hiprandState *state) {
    int de = formiga->cidadeAtual;
    double denom = 0.0;
    for (int para = 0; para < numCidades; ++para) {
        if (formiga->visitado[para] == 0) {
            denom += pow(feromonios[de * numCidades + para], ALFA) * pow(1.0 / distancias[de * numCidades + para], BETA);
        }
    }

    // Gerar um número aleatório entre 0 e 1
    double limite = (double)(hiprand(state) % 1000000000) / 1000000000.0;
    double probAcumulada = 0.0;
    // Calcular a probabilidade acumulada e selecionar a próxima cidade
    for (int para = 0; para < numCidades; ++para) {
        if (formiga->visitado[para] == 0) {
            probAcumulada += pow(feromonios[de * numCidades + para], ALFA) * pow(1.0 / distancias[de * numCidades + para], BETA) / denom;
            if (probAcumulada >= limite) {
                return para;
            }
        }
    }

    // Fallback para garantir que uma cidade não visitada seja escolhida
    for (int para = 0; para < numCidades; ++para) {
        if (formiga->visitado[para] == 0) {
            return para;
        }
    }
    return -1; 
}
